//THE FOLLOWING PROGRAM in the current version can support only the use of one GPU

#include <stdio.h>
#include <string.h>
#include <stdbool.h> 
#include <stdlib.h>

//apparently this is needed for cuda ON WINDOWS
#include <hip/hip_runtime.h>


#include <hip/driver_types.h>

//for the clauses
#define UNSATISFIED -2
#define SATISFIED 2

//for unit propagation
#define CONFLICT -3
#define NO_CONFLICT 3

//for the problem
#define SATISFIABLE true
#define UNSATISFIABLE false

//for the literals in the matrix
#define NEGATED_LIT -1
#define POSITIVE_LIT 1
#define NO_LIT 0
//for the literals in the partial assignment
#define UNASSIGNED 0
#define TRUE 1
#define FALSE -1

//for the pure literal check
#define FIRST_APPEARENCE 0
#define APPEARS_ONLY_POS 1
#define APPEARS_ONLY_NEG -1
#define APPEARS_BOTH 3


struct NoGoodDataC {
    int currentNoGoods; //the number of non satisfied clauses (yet)
    int varsYetToBeAssigned; //the number of variables that are not yet assigned
    int* partialAssignment;//we skip the cell 0 in order to maintain coherence with the var numbering
    int* noOfVarPerNoGood; //a int array that holds the number of variables in each clause
    int* lonelyVar; //a int array that holds if noOfVarPerNoGood[i]==1 the index of the only variable in the clause  
};

void readFile_allocateMatrix(const char*, struct NoGoodDataC*);
void printError(char*);
void popualteMatrix(FILE*, struct NoGoodDataC*);
void allocateMatrix();
void deallocateMatrix();
__global__ void pureLiteralCheck(int*, int *, int *);
__global__ void removeNoGoodSetsContaining(int*, int*, int*,int *);
//__global__ void unitPropagation(int* dev_matrix,int * dev_partialAssignment,int * dev_varBothNegatedAndNot, int * dev_noOfVarPerNoGood,int* dev_lonelyVar);
//__device__ void removeLiteralFromNoGoods(int ,int**, int, int,int*,int*,int*, int*);

void printMatrix(int**);
//algorithm data:

int** matrix; //the matrix that holds the clauses
int* dev_matrix; //the matrix that holds the clauses on the device

int noVars = -1; //the number of vars in the model
__device__ int dev_noVars; //the number of vars in the model on the device

int noNoGoods = -1; //the no of clauses (initially) in the model
__device__ int dev_noNoGoods; //the no of clauses (initially) in the model on the device

//create the strucure
struct NoGoodDataC data;
__device__ int dev_currentNoGoods; //the number of non satisfied clauses (yet)
__device__ int dev_varsYetToBeAssigned; //the number of variables that are not yet assigned
int* dev_partialAssignment;//we skip the cell 0 in order to maintain coherence with the var numbering
int* dev_noOfVarPerNoGood; //a int array that holds the number of variables in each clause
int* dev_lonelyVar; //a int array that holds if noOfVarPerNoGood[i]==1 the index of the only variable in the clause  

int* varBothNegatedAndNot = NULL; //a int array that holds the status of the variables in the clauses (see the defines above)
int* dev_varBothNegatedAndNot=NULL;
bool breakSearchAfterOne = false; //if true, the search will stop after the first solution is found
bool solutionFound = false; //if true, a solution was found, used to stop the search

//technical (GPU related) data:
struct hipDeviceProp_t deviceProperties; //on WINDOWS it seems we need to add the "Struct" 
int noOfVarsPerThread = 4; //the number of variables that each thread will handle in unit propagation, so that each thread will deal with 32 byte of memory (1 mem. transfer)
__device__ int dev_noOfVarsPerThread = 4;
__device__ int dev_noNoGoodsperThread=2;
__device__ int dev_conflict;
int conflict=NO_CONFLICT;

int main(int argc, char const* argv[]) {

    int GPUSno;

    //we just check, then GPUSno won't be used to scale the program
    if (hipGetDeviceCount(&GPUSno) != hipSuccess) {
		//printError("No GPU detected");
		return -1;
	}

    //if the user didn't insert the file path or typed more
    if (argc != 2) {
        //printError("Insert the file path");
        return -2;
    }

    hipGetDeviceProperties(&deviceProperties,0);
    printf("The detected GPU has %d SMs", deviceProperties.multiProcessorCount);

    //we populate it with the data from the file
    readFile_allocateMatrix(argv[1], &data);
    //printMatrix(matrix);
    //now we have the matrix on the device,alongside dev_data and dev_noVars & dev_noNoGoods 

    //allocate the partial assignment array
  
    //printMatrix(matrix);

    int blocksToLaunch = 1;// (int) noVars / (noOfVarsPerThread*128);
    //thus we launch the number of blocks needed, each thread will handle noOfVarsPerThread variables (128 threads per block, four warps)
    //dev_noNoGoodsperThread=(int)(noNoGoods/blocksToLaunch*32);
    //ths for the  vars
    pureLiteralCheck<<<blocksToLaunch,32>>>(dev_matrix,dev_partialAssignment, dev_varBothNegatedAndNot);
 	//ths for the noGoods
    removeNoGoodSetsContaining<<<blocksToLaunch,32>>>(dev_matrix,dev_partialAssignment, dev_varBothNegatedAndNot,dev_varBothNegatedAndNot);
   
    //cudaDeviceSynchronize(); su unico stream non  serve
    conflict=NO_CONFLICT;
	hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(dev_conflict), &conflict, sizeof(int), 0, hipMemcpyHostToDevice);

    //printf("cose%s\n",cudaGetErrorString(err) );

	//copio la mat
	for (int i = 0; i < noNoGoods; i++) {
        hipMemcpy(matrix[i], dev_matrix + i * ((noVars + 1)), (noVars + 1) * sizeof(int), hipMemcpyDeviceToHost);
    }
	


    //unitPropagation<<<blocksToLaunch,32>>>(dev_matrix,dev_partialAssignment, dev_varBothNegatedAndNot,dev_noOfVarPerNoGood, dev_lonelyVar); 
    //if we find a conlfict at the top level, the problem is unsatisfiable
    hipDeviceSynchronize();
    err=hipMemcpyFromSymbol(&conflict, HIP_SYMBOL(dev_conflict), sizeof(int), 0, hipMemcpyDeviceToHost);
	//printf("cose%s\n",cudaGetErrorString(err) );
    hipMemcpy((data.partialAssignment), dev_partialAssignment, (noVars + 1)*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=1;i< noVars+1; i++){
		printf("el: %d\n",(data.partialAssignment[i]) );
	}
    if (conflict==CONFLICT) {
        printf("\n\n\n**********UNSATISFIABLE**********\n\n\n");
        //deallocateMatrix(&(data.matrix));
        return -1;
    }
	/*
    //if we somehow already have an assignment, we can skip the search
    if (data.currentNoGoods == 0) {
        printf("\n\n\n**********SATISFIABLE**********\n\n\n");
        printf("Assignment:\n");
        printVarArray(data.partialAssignment);
        deallocateMatrix(&(data.matrix));
        return;
    }

    //we choose a variable and we start the search
    int varToAssign = chooseVar(data.partialAssignment);

    if (solve(data, varToAssign, TRUE) || solve(data, varToAssign, FALSE)) {
        printf("\n\n\n**********SATISFIABLE**********\n\n\n");
    }
    else {
        printf("\n\n\n**********UNSATISFIABLE**********\n\n\n");
    }
    */
  //  deallocateMatrix(&(data.matrix));
    for (int i = 0; i < noNoGoods; i++) {
        hipMemcpy(matrix[i], dev_matrix + i * ((noVars + 1)), (noVars + 1) * sizeof(int), hipMemcpyDeviceToHost);
    }

    printMatrix(matrix);

    return 1;
}

//reads the content of a simil DMACS file and populates the data structure
// (not the fanciest function but it's called just once)
void readFile_allocateMatrix(const char* str, struct NoGoodDataC* data) {

    FILE* ptr;
    char ch;
    ptr = fopen(str, "r");

    if (NULL == ptr) {
        //printError("No such file or can't be opened");
        return;
    }
    bool isComment = true;
    bool newLine = true;
    while (isComment == true && !feof(ptr)) {
        ch = fgetc(ptr);

        //a comment
        if (ch == 'c' && newLine == true) {
            isComment = true;
        }
        if (ch == 'p' && newLine == true) {
            isComment = false;
        }

        if (ch == '\n') {
            newLine = true;
        }
        else {
            newLine = false;
        }
    }

    //skip over "p nogood"
    int i = 8;
    while (!feof(ptr) && i > 0) {
        ch = fgetc(ptr);
        i--;
    }

    //ignore return value for now
    fscanf(ptr, "%d", &noVars);
    fscanf(ptr, "%d", &noNoGoods);
    printf("\nnumber of vars: %d \n", noVars);
    printf("number of nogoods: %d \n", noNoGoods);

    hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(dev_noNoGoods), &noNoGoods, sizeof(int), 0, hipMemcpyHostToDevice);
    //printf("copy No goods%s\n",cudaGetErrorString(err) );
    err=hipMemcpyToSymbol(HIP_SYMBOL(dev_noVars), &noVars, sizeof(int), 0, hipMemcpyHostToDevice);
	//printf("copy No vars%s\n",cudaGetErrorString(err) );

    data->currentNoGoods = noNoGoods;
    data->varsYetToBeAssigned = noVars;   

    err=hipMemcpyToSymbol(HIP_SYMBOL(dev_currentNoGoods), &noNoGoods, sizeof(int), 0, hipMemcpyHostToDevice);
    //printf("copy current No goods%s\n",cudaGetErrorString(err) );
    err=hipMemcpyToSymbol(HIP_SYMBOL(dev_varsYetToBeAssigned), &noVars, sizeof(int), 0, hipMemcpyHostToDevice);
    //printf("copy current No vars%s\n",cudaGetErrorString(err) );
    popualteMatrix(ptr, data);
    fclose(ptr);
}


//subprocedure called by readFile_allocateMatrix it populates the data structure and other arrays such as varBothNegatedAndNot
void popualteMatrix(FILE* ptr, struct NoGoodDataC* data) {

    allocateMatrix();

    varBothNegatedAndNot = (int*)calloc(noVars + 1, sizeof(int));
    hipError_t err=hipMalloc((void **)&dev_varBothNegatedAndNot, (noVars + 1) * sizeof(int));
	//printf("allocated varBothNegatedAndNot %s\n",cudaGetErrorString(err) );
    data->noOfVarPerNoGood = (int*)calloc(noNoGoods, sizeof(int));

    data->lonelyVar = (int*)calloc(noNoGoods, sizeof(int));
    data->partialAssignment = (int*)calloc(noVars + 1, sizeof(int));
    err=hipMalloc((void**)&dev_partialAssignment, (noVars + 1) * sizeof(int));
    //printf("allocated dev_partialAssignment %s\n",cudaGetErrorString(err) );
    err=hipMalloc((void**)&dev_noOfVarPerNoGood, noNoGoods* sizeof(int));
	//printf("allocated dev_noOfVarPerNoGood %s\n",cudaGetErrorString(err) );
    err=hipMalloc((void**)&dev_lonelyVar, noNoGoods * sizeof(int));

	//printf("allocated dev_lonelyVar %s\n",cudaGetErrorString(err) );


    for (int i = 0; i < noVars + 1; i++) {
        varBothNegatedAndNot[i] = FIRST_APPEARENCE;
    }

    int clauseCounter = 0;
    int literal = 0;
    while (!feof(ptr) && clauseCounter < noNoGoods) {

        //no idea why fscanf READS positive number as negative and vv (on Windows) 
        fscanf(ptr, "%d", &literal);
        if (literal == 0) {
            matrix[clauseCounter][0] = UNSATISFIED; //the first cell of the matrix is the status of the clause
            clauseCounter++;
        } else {

            int sign = literal > 0 ? POSITIVE_LIT : NEGATED_LIT;
            matrix[clauseCounter][literal * sign] = sign;
            data->noOfVarPerNoGood[clauseCounter]++;
            //if i have more vars i won't read this, so it can contain a wrong value (if the literal is just one the value will be correct)
            data->lonelyVar[clauseCounter] = literal * sign;

            //populate the varBothNegatedAndNot array
            if (varBothNegatedAndNot[literal * sign] == FIRST_APPEARENCE)
                varBothNegatedAndNot[literal * sign] = sign;
            if (varBothNegatedAndNot[literal * sign] == APPEARS_ONLY_POS && sign == NEGATED_LIT)
                varBothNegatedAndNot[literal * sign] = APPEARS_BOTH;
            if (varBothNegatedAndNot[literal * sign] == APPEARS_ONLY_NEG && sign == POSITIVE_LIT)
                varBothNegatedAndNot[literal * sign] = APPEARS_BOTH;
        }
    }
    //we now copy the content of the matrix to the device (https://forums.developer.nvidia.com/t/passing-dynamically-allocated-2d-array-to-device/43727 apparenlty works just for static matrices)
 	for(int i = 0; i < noNoGoods; i++) {
		hipError_t err= hipMemcpy((dev_matrix+i* ((noVars + 1) )), matrix[i], (noVars + 1) * sizeof(int), hipMemcpyHostToDevice);
        //printf("%s\n", cudaGetErrorString(err));
    }

    //we copy varBothNegatedAndNot
    err= hipMemcpy(dev_varBothNegatedAndNot , varBothNegatedAndNot, sizeof(int)* (noVars + 1), hipMemcpyHostToDevice);
  	//printf("copied dev_varBothNegatedAndNot %s\n",cudaGetErrorString(err) );
    //and the other vectors
    //printf("%d\n", noVars+1 );
    err = hipMemcpy(dev_partialAssignment, (data->partialAssignment), sizeof(int) * (noVars + 1), hipMemcpyHostToDevice);
   
    
    //printf("copied dev_partialAssignment %s\n",cudaGetErrorString(err) );
    err = hipMemcpy(dev_noOfVarPerNoGood, data->noOfVarPerNoGood, sizeof(int) * (noNoGoods), hipMemcpyHostToDevice);
    
    
    err = hipMemcpy(dev_lonelyVar, data->lonelyVar, sizeof(int) * (noNoGoods), hipMemcpyHostToDevice);
    //printf("copied dev_lonelyVar %s\n",cudaGetErrorString(err) );
    
}

//prints str with "ERROR" in front of it
void printError(char* str) {
    printf("ERROR: %s \n", str);
}

//allocates the matrix
void allocateMatrix() {
    matrix = (int**)calloc(noNoGoods, sizeof(int*));
    //indeed arrays of pointers are not a good idea on the GPU
    hipError_t err=hipMalloc((void **) &dev_matrix, noNoGoods * (noVars + 1) * sizeof(int));

    //printf("allocated matrix %s\n",cudaGetErrorString(err) );
    for (int i = 0; i < noNoGoods; i++) {
        matrix[i] = (int*)calloc(noVars + 1, sizeof(int));
    }

}

//deallocates the matrix
void deallocateMatrix() {

    for (int i = 0; i < noNoGoods; i++) {
        free(matrix[i]);
    }
    free(matrix);
    hipFree(&dev_matrix);
}

//removes the literal (by assigning a value) from the no goods IF it's UNASSIGNED and shows up with only one sign (in the remaining no goods)
//one th per var
__global__ void  pureLiteralCheck(int* dev_matrix,int * dev_partialAssignment,int * dev_varBothNegatedAndNot) {
 
    int thPos = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("Th. %d\n",thPos);
    //we scan each var (ths deal with vars)
    for (int i = thPos*dev_noOfVarsPerThread; i < thPos*dev_noOfVarsPerThread + dev_noOfVarsPerThread; i++) {
    	if (i<=dev_noVars && dev_partialAssignment[i] == UNASSIGNED && (dev_varBothNegatedAndNot[i] == APPEARS_ONLY_POS || dev_varBothNegatedAndNot[i] == APPEARS_ONLY_NEG)) {
	    	
	    	//printf("th. no %d working on pos %d \n",thPos,i);
	        dev_partialAssignment[i] = -dev_varBothNegatedAndNot[i];
	        //printf("th. no %d assigning to var %d\n",thPos,dev_varBothNegatedAndNot[i]);
	        //TODO substiture with one decrement at the end (e.g. warp level reduction)
	        atomicAdd(&dev_varsYetToBeAssigned,-1);
	        
	        //this can't be called here, it would need too much serialization
	        //removeNoGoodSetsContaining(i,&(dev_matrix), &(dev_currentNoGoods), dev_varBothNegatedAndNot[i]);
    	}
    	__syncthreads();
    }   
   // }
}

//removes (assigns 'falsified' satisfied) the no goods if they contain the literal varIndex with the indicated sign

//one th per no good
__global__ void removeNoGoodSetsContaining(int* matrix, int* currentNoGoods, int* dev_varBothNegatedAndNot,int* sing) {
	
	int thPos = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("th %d in, lookin' %d \n",thPos,*(*matrix+thPos));   
	//printf("th %d dev_noNoGoods %d \n",thPos,dev_noNoGoods);         
    //we scan each no_good (each th scans reading the first cell of matrix and the relative var pos)
    
    int decrease=0;
    for (int i = thPos*dev_noNoGoodsperThread; i <= thPos*dev_noNoGoodsperThread + dev_noNoGoodsperThread; i++) {
	    if (i<dev_noNoGoods){
	    	//fixed a no good (thread) we loop on the row of the matrix (in this way ONLY ONE thead access each cell of the first column)
	    	for(int varIndex=1; varIndex<=dev_noVars; varIndex++){
		    	if(*(matrix+i*(dev_noVars+1)+varIndex) == sing[varIndex] && (*matrix + i*(dev_noVars+1)) != SATISFIED) {
			        //remove the nogood set
			        printf("th. no %d changing  %d of clause to SAT \n",thPos,*(matrix + thPos*(dev_noVars+1)));
			        *(matrix + i*(dev_noVars+1)) = SATISFIED; //VA FATTA ATOMIC
			      
			        //TODO substiture with one decrement at the end (e.g. warp level reduction)
			       	decrease--;
			        //(*currentNoGoods)--;
			    }
			    __syncthreads();
		    }
		}
    }
    atomicAdd(currentNoGoods, decrease);
}
//prints the content of the matrix (the first column is the status of each clause)
void printMatrix(int** matrix) {
    printf("\n");
    for (int i = 0; i < noNoGoods; i++) {
        if (matrix[i][0] == UNSATISFIED)
            printf("UNSATISFIED ");
        else
            printf("SATISFIED   ");
        for (int j = 1; j < noVars + 1; j++) {
            if (matrix[i][j] < 0)
                printf("%d ", matrix[i][j]);
            else
                printf(" %d ", matrix[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}
/*
__global__ void unitPropagation(int* dev_matrix,int * dev_partialAssignment,int * dev_varBothNegatedAndNot,int *dev_noOfVarPerNoGood,int* dev_lonelyVar) {
    
    int thPos = blockIdx.x * blockDim.x + threadIdx.x;
    int removeLiteralFromNoGoodsRETURN;

    //for each no good (th deals with ng)
    for (int i = thPos*dev_noOfVarPerNoGood; i < thPos*dev_noOfVarPerNoGood + dev_noOfVarPerNoGood; i++) {

    	//printf("th %d lookin at cell %d \n",thPos,i );
    	if(dev_conflict!=NO_CONFLICT){
    		printf("someone said conflict\n");
    		if(threadIdx.x==0){
    			printf("Hi, th 0 setting conflict\n");
    			dev_conflict=CONFLICT;
    		}
    		__syncthreads();
    		break;
    	}
    	if(i<=dev_noNoGoods && *(dev_matrix+i*(dev_noVars+1)) == UNSATISFIED && dev_noOfVarPerNoGood[i] == 1){
            printf("th %d inside at no good: %d, seing: %d\n",thPos,i,*(dev_matrix+i*(dev_noVars+1)));
            //lonelyVar[i] is a column index
            dev_partialAssignment[dev_lonelyVar[i]] = *(dev_matrix+i*(dev_noVars+1)+dev_lonelyVar[i]) > 0 ? FALSE : TRUE;
            atomicAdd(&dev_varsYetToBeAssigned,-1);

            //we update the status of the no good
            removeNoGoodSetsContaining(i,&(dev_matrix), &(dev_currentNoGoods), -(dev_partialAssignment[dev_lonelyVar[i]]));
          
            removeLiteralFromNoGoods(i,&(dev_matrix), dev_lonelyVar[i], dev_partialAssignment[dev_lonelyVar[i]],dev_noOfVarPerNoGood,dev_partialAssignment,dev_lonelyVar ,&removeLiteralFromNoGoodsRETURN);
            if(removeLiteralFromNoGoodsRETURN==CONFLICT){
            	atomicAdd(&dev_conflict,1);
            	printf("FOUND, i's should be all equal to: %d",i);
            }
        }
        __syncthreads();

    }

}

//a call deals with one no_good
__device__ void removeLiteralFromNoGoods(int i,int** dev_matrix, int varIndex, int sign,int *dev_noOfVarPerNoGood,int* dev_partialAssignment,int* dev_lonelyVar, int* returnV) {
    //scan column (varIndex) of matrix, we want to skip the first column, so i%(dev_noVars+1)!=0
   if (*(*dev_matrix+i+varIndex) == sign) {
        //remove the literal
        atomicAdd((dev_noOfVarPerNoGood+i), -1);
        if (dev_noOfVarPerNoGood[i] == 1) {
            //search and assing the literal to the lonelyVar
            for (int j = 1; j < dev_noVars + 1; j++) {
            	//fix if first condition
                if (*(*dev_matrix+i+j) != NO_LIT && dev_partialAssignment[j] == UNASSIGNED) {
                    dev_lonelyVar[i] = j;
                }
                __syncthreads();
            }
        }        
    }
    __syncthreads();
    if(dev_noOfVarPerNoGood[i] == 0) 
        *(returnV) = CONFLICT;
    else
    //dev_noOfVarPerNoGood[i]-3 //se era 0 ritorno conflict
    	*(returnV) = NO_CONFLICT;
}*/
